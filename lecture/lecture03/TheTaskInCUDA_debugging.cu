#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
# include <stdio.h>
# include <hip/hip_runtime.h>
# include <>
# include "TheEmployeesSalary.h"
# include <stdlib.h>
 
hipError_t thehelperfunction(){
    
}

// global kernel function -> which can only access host function
__global__ void TaskDoer(const double* array, double* newSalaries, int size)
{
    // 병렬화하려면 우선 thread index를 가져와야 함.
    // int ID = blockIdx.x * blockDim.x +  threadIdx.x;
    // newSalaries[ID] = array[ID] + (array[ID] * 15 / 100) + 5000;

    // thread index가 SIZE를 초기화하면? ->
    int ID = blockIdx.x * blockDim.x +  threadIdx.x;
    if (ID < size){
        newSalaries[ID] = array[ID] * 1.15 + 5000.0;
    }

}

int main()
{
    int size = sizeof(TheArrayOfSalaries) / sizeof(TheArrayOfSalaries[0]);
    // device 는 host pointer, host variable에 직접 access 불가능
    // variable 할당
    double* d_Array;
    double* d_NewSalaries;
    double* newSalaries;

    // memory 할당
    hipMalloc((void**)& d_Array, size * sizeof(double)); // 장치에 8byte memory 할당 + 배열이므로 *100
    hipMalloc((void**)& d_NewSalaries, size * sizeof(double));

    hipMemcpy(d_Array, TheArrayOfSalaries, size * sizeof(double), hipMemcpyHostToDevice);
    // 사용할 thread 수 지정
    int threadsPerBlock=256;
    int blocksPerGrid=(size + threadsPerBlock - 1) / threadsPerBlock;
    
    TaskDoer<<<blocksPerGrid, threadsPerBlock>>>(d_Array, d_NewSalaries, size);
    hipDeviceSynchronize();
    newSalaries = (double*)malloc(size * sizeof(double));
    hipMemcpy(newSalaries, d_NewSalaries, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i ++)
    {
        printf("%f\n", newSalaries[i]);
    }
    free(newSalaries);
    hipFree(d_Array);
    hipFree(d_NewSalaries);
    return 0;
}